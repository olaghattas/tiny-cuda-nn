#include "hip/hip_runtime.h"
#include <tiny-cuda-nn/gpu_memory.h>

//TCNN_NAMESPACE_BEGIN

/// Managed memory on the Device
template<typename T>
tcnn::GPUMemory<T>::GPUMemory() {}

template<typename T>
tcnn::GPUMemory<T>::GPUMemory(size_t size, bool managed) : m_managed{managed} {
    resize(size);
}

template<typename T>
tcnn::GPUMemory<T> &tcnn::GPUMemory<T>::operator=(GPUMemory<T> &&other) {
    std::swap(m_data, other.m_data);
    std::swap(m_size, other.m_size);
    std::swap(m_managed, other.m_managed);
    return *this;
}

template<typename T>
tcnn::GPUMemory<T>::GPUMemory(GPUMemory<T> &&other) {
    *this = std::move(other);
}

// Don't permit copy assignment to prevent performance accidents.
// Copy is permitted through an explicit copy constructor.
template<typename T>
tcnn::GPUMemory<T>::GPUMemory(const GPUMemory<T> &other) {
    m_managed = other.managed();
    copy_from_device(other);
}

template<typename T>
void tcnn::GPUMemory<T>::check_guards() const {
#if DEBUG_GUARD_SIZE > 0
    if (!m_data)
        return;
    uint8_t buf[DEBUG_GUARD_SIZE];
    const uint8_t *rawptr=(const uint8_t *)m_data;
    hipMemcpy(buf, rawptr-DEBUG_GUARD_SIZE, DEBUG_GUARD_SIZE, hipMemcpyDeviceToHost);
    for (int i=0;i<DEBUG_GUARD_SIZE;++i) if (buf[i] != 0xff) {
        printf("TRASH BEFORE BLOCK offset %d data %p, read 0x%02x expected 0xff!\n", i, m_data, buf[i] );
        break;
    }
    hipMemcpy(buf, rawptr+m_size*sizeof(T), DEBUG_GUARD_SIZE, hipMemcpyDeviceToHost);
    for (int i=0;i<DEBUG_GUARD_SIZE;++i) if (buf[i] != 0xfe) {
        printf("TRASH AFTER BLOCK offset %d data %p, read 0x%02x expected 0xfe!\n", i, m_data, buf[i] );
        break;
    }
#endif
}

template<typename T>
void tcnn::GPUMemory<T>::allocate_memory(size_t n_bytes) {
    if (n_bytes == 0) {
        return;
    }

#ifdef TCNN_VERBOSE_MEMORY_ALLOCS
    std::cout << "GPUMemory: Allocating " << bytes_to_string(n_bytes) << "." << std::endl;
#endif

    uint8_t *rawptr = nullptr;
    if (m_managed) {
        CUDA_CHECK_THROW(hipMallocManaged(&rawptr, n_bytes + DEBUG_GUARD_SIZE * 2));
    } else {
        CUDA_CHECK_THROW(hipMalloc(&rawptr, n_bytes + DEBUG_GUARD_SIZE * 2));
    }
#if DEBUG_GUARD_SIZE > 0
    CUDA_CHECK_THROW(hipMemset(rawptr, 0xff, DEBUG_GUARD_SIZE));
    CUDA_CHECK_THROW(hipMemset(rawptr + n_bytes + DEBUG_GUARD_SIZE, 0xfe, DEBUG_GUARD_SIZE));
#endif
    if (rawptr) rawptr += DEBUG_GUARD_SIZE;
    m_data = (T *) (rawptr);
    total_n_bytes_allocated() += n_bytes;
}

template<typename T>
void tcnn::GPUMemory<T>::free_memory() {
    if (!m_data) {
        return;
    }

    uint8_t *rawptr = (uint8_t *) m_data;
    if (rawptr) rawptr -= DEBUG_GUARD_SIZE;
    CUDA_CHECK_THROW(hipFree(rawptr));

    total_n_bytes_allocated() -= get_bytes();

    m_data = nullptr;
    m_size = 0;
}

/// Frees memory again
template<typename T>
TCNN_HOST_DEVICE tcnn::GPUMemory<T>::~GPUMemory() {
#ifndef __CUDA_ARCH__
    try {
        if (m_data) {
            free_memory();
            m_size = 0;
        }
    } catch (std::runtime_error error) {
        // Don't need to report on memory-free problems when the driver is shutting down.
        if (std::string{error.what()}.find("driver shutting down") == std::string::npos) {
            std::cerr << "Could not free memory: " << error.what() << std::endl;
        }
    }
#endif
}

/** @name Resizing/enlargement
 *  @{
 */
/// Resizes the array to the exact new size, even if it is already larger
template<typename T> void tcnn::GPUMemory<T>::resize(const size_t size) {
    if (m_size != size) {
        if (m_size) {
            try {
                free_memory();
            } catch (std::runtime_error error) {
                throw std::runtime_error{fmt::format("Could not free memory: {}", error.what())};
            }
        }

        if (size > 0) {
            try {
                allocate_memory(size * sizeof(T));
            } catch (std::runtime_error error) {
                throw std::runtime_error{fmt::format("Could not allocate memory: {}", error.what())};
            }
        }

        m_size = size;
    }
}

/// Enlarges the array if its size is smaller
template<typename T>
void tcnn::GPUMemory<T>::enlarge(const size_t size) {
    if (size > m_size) {
        resize(size);
    }
}
/** @} */

/** @name Memset
 *  @{
 */
/// Sets the memory of the first num_elements to value
template<typename T>
void tcnn::GPUMemory<T>::memset(const int value, const size_t num_elements, const size_t offset) {
    if (num_elements + offset > m_size) {
        throw std::runtime_error{
                fmt::format("Could not set memory: Number of elements {}+{} larger than allocated memory {}.",
                            num_elements, offset, m_size)};
    }

    CUDA_CHECK_THROW(hipMemset(m_data + offset, value, num_elements * sizeof(T)));
}

/// Sets the memory of the all elements to value
template<typename T>
void tcnn::GPUMemory<T>::memset(const int value) {
    memset(value, m_size);
}
/** @} */

/** @name Copy operations
 *  @{
 */
/// Copy data of num_elements from the raw pointer on the host
template<typename T>
void tcnn::GPUMemory<T>::copy_from_host(const T *host_data, const size_t num_elements) {
    CUDA_CHECK_THROW(hipMemcpy(data(), host_data, num_elements * sizeof(T), hipMemcpyHostToDevice));
}

/// Copy num_elements from the host vector
template<typename T>
void tcnn::GPUMemory<T>::copy_from_host(const std::vector<T> &data, const size_t num_elements) {
    if (data.size() < num_elements) {
        throw std::runtime_error{
                fmt::format("Trying to copy {} elements, but vector size is only {}.", num_elements, data.size())};
    }
    copy_from_host(data.data(), num_elements);
}

/// Copies data from the raw host pointer to fill the entire array
template<typename T>
void tcnn::GPUMemory<T>::copy_from_host(const T *data) {
    copy_from_host(data, m_size);
}

/// Copies num_elements of data from the raw host pointer after enlarging the array so that everything fits in
template<typename T>
void tcnn::GPUMemory<T>::enlarge_and_copy_from_host(const T *data, const size_t num_elements) {
    enlarge(num_elements);
    copy_from_host(data, num_elements);
}

/// Copies num_elements from the host vector after enlarging the array so that everything fits in
template<typename T>
void tcnn::GPUMemory<T>::enlarge_and_copy_from_host(const std::vector<T> &data, const size_t num_elements) {
    enlarge_and_copy_from_host(data.data(), num_elements);
}

/// Copies the entire host vector after enlarging the array so that everything fits in
template<typename T>
void tcnn::GPUMemory<T>::enlarge_and_copy_from_host(const std::vector<T> &data) {
    enlarge_and_copy_from_host(data.data(), data.size());
}

/// Copies num_elements of data from the raw host pointer after resizing the array
template<typename T>
void tcnn::GPUMemory<T>::resize_and_copy_from_host(const T *data, const size_t num_elements) {
    resize(num_elements);
    copy_from_host(data, num_elements);
}

/// Copies num_elements from the host vector after resizing the array
template<typename T>
void tcnn::GPUMemory<T>::resize_and_copy_from_host(const std::vector<T> &data, const size_t num_elements) {
    resize_and_copy_from_host(data.data(), num_elements);
}

/// Copies the entire host vector after resizing the array
template<typename T>
void tcnn::GPUMemory<T>::resize_and_copy_from_host(const std::vector<T> &data) {
    resize_and_copy_from_host(data.data(), data.size());
}

/// Copies the entire host vector to the device. Fails if there is not enough space available.
template<typename T>
void tcnn::GPUMemory<T>::copy_from_host(const std::vector<T> &data) {
    if (data.size() < m_size) {
        throw std::runtime_error{
                fmt::format("Trying to copy {} elements, but vector size is only {}.", m_size, data.size())};
    }
    copy_from_host(data.data(), m_size);
}

/// Copies num_elements of data from the raw host pointer to the device. Fails if there is not enough space available.
template<typename T>
void tcnn::GPUMemory<T>::copy_to_host(T *host_data, const size_t num_elements) const {
    if (num_elements > m_size) {
        throw std::runtime_error{
                fmt::format("Trying to copy {} elements, but memory size is only {}.", num_elements, m_size)};
    }

    CUDA_CHECK_THROW(hipMemcpy(host_data, data(), num_elements * sizeof(T), hipMemcpyDeviceToHost));
}

/// Copies num_elements from the device to a vector on the host
template<typename T>
void tcnn::GPUMemory<T>::copy_to_host(std::vector<T> &data, const size_t num_elements) const {
    if (data.size() < num_elements) {
        throw std::runtime_error{
                fmt::format("Trying to copy {} elements, but vector size is only {}.", num_elements, data.size())};
    }

    copy_to_host(data.data(), num_elements);
}

/// Copies num_elements from the device to a raw pointer on the host
template<typename T>
void tcnn::GPUMemory<T>::copy_to_host(T *data) const {
    copy_to_host(data, m_size);
}

/// Copies all elements from the device to a vector on the host
template<typename T>
void tcnn::GPUMemory<T>::copy_to_host(std::vector<T> &data) const {
    if (data.size() < m_size) {
        throw std::runtime_error{
                fmt::format("Trying to copy {} elements, but vector size is only {}", m_size, data.size())};
    }

    copy_to_host(data.data(), m_size);
}

/// Copies size elements from another device array to this one, automatically resizing it
template<typename T>
void tcnn::GPUMemory<T>::copy_from_device(const tcnn::GPUMemory<T> &other, const size_t size) {
    if (size == 0) {
        return;
    }

    if (m_size < size) {
        resize(size);
    }

    CUDA_CHECK_THROW(hipMemcpy(m_data, other.m_data, size * sizeof(T), hipMemcpyDeviceToDevice));
}

/// Copies data from another device array to this one, automatically resizing it
template<typename T>
void tcnn::GPUMemory<T>::copy_from_device(const tcnn::GPUMemory<T> &other) {
    copy_from_device(other, other.m_size);
}

// Created an (owned) copy of the data
template<typename T>
tcnn::GPUMemory<T> tcnn::GPUMemory<T>::copy(size_t size) const {
    GPUMemory<T> result{size};
    result.copy_from_device(*this);
    return result;
}

template<typename T>
tcnn::GPUMemory<T> tcnn::GPUMemory<T>::copy() const {
    return copy(m_size);
}

template<typename T>
T *tcnn::GPUMemory<T>::data() const {
    check_guards();
    return m_data;
}

template<typename T>
bool tcnn::GPUMemory<T>::managed() const {
    return m_managed;
}

template<typename T>
T &tcnn::GPUMemory<T>::at(size_t idx) const {
    if (!m_managed) {
        throw std::runtime_error{fmt::format("GPUMemory::at() not permitted if not managed.")};
    }

    if (idx > m_size) {
        throw std::runtime_error{fmt::format("GPUMemory our of bounds: idx={} size={}", idx, m_size)};
    }

    return m_data[idx];
}

template<typename T>
TCNN_HOST_DEVICE T &tcnn::GPUMemory<T>::operator[](size_t idx) const {
#ifdef DEBUG_BUFFER_OVERRUN
    if (idx > m_size) {
        printf("WARNING: buffer overrun of %p at idx %zu\n", idx);
    }
#endif
    return m_data[idx];
}

template<typename T>
TCNN_HOST_DEVICE T &tcnn::GPUMemory<T>::operator[](uint32_t idx) const {
#ifdef DEBUG_BUFFER_OVERRUN
    if (idx > m_size) {
        printf("WARNING: buffer overrun of %p at idx %u\n", idx);
    }
#endif
    return m_data[idx];
}

template<typename T>
size_t tcnn::GPUMemory<T>::get_num_elements() const {
    return m_size;
}

template<typename T>
size_t tcnn::GPUMemory<T>::size() const {
    return get_num_elements();
}

template<typename T>
size_t tcnn::GPUMemory<T>::get_bytes() const {
    return m_size * sizeof(T);
}

template<typename T>
size_t tcnn::GPUMemory<T>::bytes() const {
    return get_bytes();
}


bool tcnn::Interval::operator<(const Interval &other) const {
    // This operator is used to sort non-overlapping intervals. Since intervals
    // may be empty, the second half of the following expression is required to
    // resolve ambiguity when `end` of adjacent empty intervals is equal.
    return end < other.end || (end == other.end && start < other.start);
}

bool tcnn::Interval::overlaps(const Interval &other) const {
    return !intersect(other).empty();
}

tcnn::Interval tcnn::Interval::intersect(const Interval &other) const {
    return {std::max(start, other.start), std::min(end, other.end)};
}

bool tcnn::Interval::valid() const {
    return end >= start;
}

bool tcnn::Interval::empty() const {
    return end <= start;
}

size_t tcnn::Interval::size() const {
    return end - start;
}


tcnn::GPUMemoryArena::GPUMemoryArena() {
    m_device = cuda_device();

    // Align memory at least by a cache line (128 bytes).
    m_alignment = (size_t) 128;
    m_max_size = next_multiple(cuda_memory_info().total, cuda_memory_granularity());

    m_free_intervals = {{0, m_max_size}};

    if (!cuda_supports_virtual_memory()) {
        // Use regular memory as fallback
        m_fallback_memory = std::make_shared<GPUMemory<uint8_t>>();

        static bool printed_warning = false;
        if (!printed_warning) {
            printed_warning = true;
            std::cout
                    << "GPUMemoryArena: Warning: GPU " << m_device << " does not support virtual memory. "
                    << "Falling back to regular allocations, which will be larger and can cause occasional stutter."
                    << std::endl;
        }
        return;
    }

    // Reserve an address range that would be sufficient for housing the entire
    // available GPU RAM (if nothing else was using the GPU). This is unlikely
    // to exhaust all available addresses (even if multiple GPUMemoryArenas are
    // used simultaneously), while also ensuring that we never exhaust the
    // reserved address range without running out of physical memory beforehand.
    CU_CHECK_THROW(hipMemAddressReserve(&m_base_address, m_max_size, 0, 0, 0));
}


tcnn::GPUMemoryArena::~GPUMemoryArena() {
    if (in_use()) {
        std::cerr << "Attempting to free memory arena while it is still in use." << std::endl;
    }

    try {
        // Make sure we're clearing the GPU memory arena on the correct device.
        int previous_device = cuda_device();
        set_cuda_device(m_device);
        ScopeGuard revert_device = {[&]() { set_cuda_device(previous_device); }};

        CUDA_CHECK_THROW(hipDeviceSynchronize());

        if (m_base_address) {
            total_n_bytes_allocated() -= m_size;

            CU_CHECK_THROW(hipMemUnmap(m_base_address, m_size));

            for (const auto &handle: m_handles) {
                CU_CHECK_THROW(hipMemRelease(handle));
            }

            CU_CHECK_THROW(hipMemAddressFree(m_base_address, m_max_size));
        }
    } catch (std::runtime_error error) {
        // Don't need to report on memory-free problems when the driver is shutting down.
        if (std::string{error.what()}.find("driver shutting down") == std::string::npos) {
            std::cerr << "Could not free memory arena: " << error.what() << std::endl;
        }
    }
}

uint8_t *tcnn::GPUMemoryArena::data() {
    return m_fallback_memory ? m_fallback_memory->data() : (uint8_t *) m_base_address;
}

std::shared_ptr<tcnn::GPUMemory<uint8_t>> tcnn::GPUMemoryArena::backing_memory() {
    return m_fallback_memory;
}

// Finds the smallest interval of free memory in the GPUMemoryArena that's
// large enough to hold the requested number of bytes. Then allocates
// that memory.
size_t tcnn::GPUMemoryArena::allocate(size_t n_bytes) {
    // Permitting zero-sized allocations is error prone
    if (n_bytes == 0) {
        n_bytes = m_alignment;
    }

    // Align allocations with the nearest cache line (at least the granularity of the memory allocations)
    n_bytes = next_multiple(n_bytes, m_alignment);

    Interval *best_candidate = &m_free_intervals.back();
    for (auto &f: m_free_intervals) {
        if (f.size() >= n_bytes && f.size() < best_candidate->size()) {
            best_candidate = &f;
        }
    }

    size_t start = best_candidate->start;

    // Note: the += operator can turn `best_candidate` into an empty interval, which is fine because it will
    // be absorbed into adjacent free intervals in later calls to `merge_adjacent_intervals`.
    m_allocated_intervals[start] = best_candidate->start += n_bytes;

    enlarge(size());

    return start;
}

void tcnn::GPUMemoryArena::free(size_t start) {
    if (m_allocated_intervals.count(start) == 0) {
        throw std::runtime_error{"Attempted to free arena memory that was not allocated."};
    }

    Interval interval = {start, m_allocated_intervals[start]};
    m_allocated_intervals.erase(start);

    m_free_intervals.insert(
            std::upper_bound(std::begin(m_free_intervals), std::end(m_free_intervals), interval),
            interval
    );

    merge_adjacent_intervals();
}

void tcnn::GPUMemoryArena::enlarge(size_t n_bytes) {
    if (n_bytes <= m_size) {
        return;
    }

    if (cuda_device() != m_device) {
        throw std::runtime_error{
                fmt::format("Attempted to use a GPUMemoryArena of device {} from the wrong device {}.", m_device,
                            cuda_device())};
    }

    if (m_fallback_memory) {
        static const double GROWTH_FACTOR = 1.5;

        CUDA_CHECK_THROW(hipDeviceSynchronize());

        m_size = next_multiple((size_t) (n_bytes * GROWTH_FACTOR), cuda_memory_granularity());
        m_fallback_memory = std::make_shared<GPUMemory<uint8_t>>(m_fallback_memory->copy(m_size));

        CUDA_CHECK_THROW(hipDeviceSynchronize());

        return;
    }

    size_t n_bytes_to_allocate = n_bytes - m_size;
    n_bytes_to_allocate = next_multiple(n_bytes_to_allocate, cuda_memory_granularity());

    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = m_device;

    m_handles.emplace_back();
    CU_CHECK_THROW(hipMemCreate(&m_handles.back(), n_bytes_to_allocate, &prop, 0));

    hipMemAccessDesc access_desc = {};
    access_desc.location.type = hipMemLocationTypeDevice;
    access_desc.location.id = prop.location.id;
    access_desc.flags = hipMemAccessFlagsProtReadWrite;

    CU_CHECK_THROW(hipMemMap(m_base_address + m_size, n_bytes_to_allocate, 0, m_handles.back(), 0));
    CU_CHECK_THROW(hipMemSetAccess(m_base_address + m_size, n_bytes_to_allocate, &access_desc, 1));
    m_size += n_bytes_to_allocate;

    total_n_bytes_allocated() += n_bytes_to_allocate;

    // Need to synchronize the device to make sure memory is available to all streams.
    if (current_capture()) {
        current_capture()->schedule_synchronize();
    } else {
        CUDA_CHECK_THROW(hipDeviceSynchronize());
    }
}

size_t tcnn::GPUMemoryArena::size() const {
    return m_free_intervals.back().start;
}

bool tcnn::GPUMemoryArena::in_use() const {
    return m_free_intervals.size() != 1 || m_free_intervals.front().size() != m_max_size;
}

tcnn::GPUMemoryArena::Allocation::Allocation(hipStream_t stream, size_t offset,
                                       const std::shared_ptr<tcnn::GPUMemoryArena> &workspace)
        : m_stream{stream}, m_data{workspace->data() + offset}, m_offset{offset}, m_workspace{workspace},
          m_backing_memory{workspace->backing_memory()} {}

tcnn::GPUMemoryArena::Allocation::~Allocation() {
    if (m_workspace) {
        m_workspace->free(m_offset);
    }
}

tcnn::GPUMemoryArena::Allocation &tcnn::GPUMemoryArena::Allocation::operator=(tcnn::GPUMemoryArena::Allocation &&other) {
    std::swap(m_stream, other.m_stream);
    std::swap(m_data, other.m_data);
    std::swap(m_offset, other.m_offset);
    std::swap(m_workspace, other.m_workspace);
    std::swap(m_backing_memory, other.m_backing_memory);
    return *this;
}

tcnn::GPUMemoryArena::Allocation::Allocation(tcnn::GPUMemoryArena::Allocation &&other) {
    *this = std::move(other);
}

uint8_t *tcnn::GPUMemoryArena::Allocation::data() {
    return m_data;
}

const uint8_t *tcnn::GPUMemoryArena::Allocation::data() const {
    return m_data;
}

hipStream_t tcnn::GPUMemoryArena::Allocation::stream() const {
    return m_stream;
}

void tcnn::GPUMemoryArena::merge_adjacent_intervals() {
    size_t j = 0;
    for (size_t i = 1; i < m_free_intervals.size(); ++i) {
        Interval &prev = m_free_intervals[j];
        Interval &cur = m_free_intervals[i];

        if (prev.end == cur.start) {
            prev.end = cur.end;
        } else {
            ++j;
            m_free_intervals[j] = m_free_intervals[i];
        }
    }
    m_free_intervals.resize(j + 1);
}


std::unordered_map<hipStream_t, std::shared_ptr<tcnn::GPUMemoryArena>> & tcnn::stream_gpu_memory_arenas() {
    static auto *stream_gpu_memory_arenas = new std::unordered_map<hipStream_t, std::shared_ptr<tcnn::GPUMemoryArena>>{};
    return *stream_gpu_memory_arenas;
}

std::unordered_map<int, std::shared_ptr<tcnn::GPUMemoryArena>> & tcnn::global_gpu_memory_arenas() {
    static auto *global_gpu_memory_arenas = new std::unordered_map<int, std::shared_ptr<tcnn::GPUMemoryArena>>{};
    return *global_gpu_memory_arenas;
}

tcnn::GPUMemoryArena::Allocation tcnn::allocate_workspace(hipStream_t stream, size_t n_bytes) {
    if (n_bytes == 0) {
        // Return a null allocation if no bytes were requested.
        return {};
    }

    auto &arena = stream ? stream_gpu_memory_arenas()[stream] : global_gpu_memory_arenas()[tcnn::cuda_device()];
    if (!arena) {
        arena = std::make_shared<tcnn::GPUMemoryArena>();
    }
    return tcnn::GPUMemoryArena::Allocation{stream, arena->allocate(n_bytes), arena};
}

size_t tcnn::align_to_cacheline(size_t bytes) {
    return next_multiple(bytes, (size_t) 128);
}

//size_t align_to_cacheline(size_t bytes) {
//    return tcnn::next_multiple(bytes, (size_t) 128);
//}

//template<typename First, typename FirstSize>
//std::tuple<First *>
//allocate_workspace_and_distribute(hipStream_t stream, tcnn::GPUMemoryArena::Allocation *alloc, size_t offset,
//                                  FirstSize first_size) {
//    *alloc = allocate_workspace(stream, offset + align_to_cacheline(first_size * sizeof(First)));
//    return std::make_tuple<First *>((First *) (alloc->data() + offset));
//}
//
//template<typename First, typename ...Types, typename FirstSize, typename ...Sizes, std::enable_if_t<
//        sizeof...(Types) != 0 && sizeof...(Types) == sizeof...(Sizes), int> >
//std::tuple<First *, Types *...>
//tcnn::allocate_workspace_and_distribute(hipStream_t stream, tcnn::GPUMemoryArena::Allocation *alloc, size_t offset,
//                                  FirstSize first_size, Sizes... sizes) {
//    auto nested = allocate_workspace_and_distribute<Types...>(stream, alloc,
//                                                              offset + align_to_cacheline(first_size * sizeof(First)),
//                                                              sizes...);
//    return std::tuple_cat(std::make_tuple<First *>((First *) (alloc->data() + offset)), nested);
//}
//
//template<typename ...Types, typename ...Sizes, std::enable_if_t<sizeof...(Types) == sizeof...(Sizes), int>>
//std::tuple<Types *...>
//tcnn::allocate_workspace_and_distribute(hipStream_t stream, GPUMemoryArena::Allocation *alloc, Sizes... sizes) {
//    return allocate_workspace_and_distribute<Types...>(stream, alloc, (size_t) 0, sizes...);
//}

void tcnn::free_gpu_memory_arena(hipStream_t stream) {
    if (stream) {
        stream_gpu_memory_arenas().erase(stream);
    } else {
        global_gpu_memory_arenas().erase(cuda_device());
    }
}

void tcnn::free_all_gpu_memory_arenas() {
    stream_gpu_memory_arenas().clear();
    global_gpu_memory_arenas().clear();
}

void annoyingFuncThatinstantiates(){
    tcnn::GPUMemory<float> tmp1;
    tcnn::GPUMemory<char> tmp2;
    tcnn::GPUMemory<half> tmp3;
    tcnn::GPUMemory<unsigned int> tmp4;
}


//TCNN_NAMESPACE_END